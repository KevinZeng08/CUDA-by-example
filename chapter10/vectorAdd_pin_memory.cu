#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

 /**
  * Vector addition: C = A + B.
  *
  * This sample is a very basic sample that implements element by element
  * vector addition. It is the same as the sample illustrating Chapter 2
  * of the programming guide with some additions like error checking.
  */

#include <stdio.h>

  // For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <omp.h>

#define START_GPU {\
hipEvent_t     start, stop;\
float   elapsedTime;\
checkCudaErrors(hipEventCreate(&start)); \
checkCudaErrors(hipEventCreate(&stop));\
checkCudaErrors(hipEventRecord(start, 0));\

#define END_GPU \
checkCudaErrors(hipEventRecord(stop, 0));\
checkCudaErrors(hipEventSynchronize(stop));\
checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop)); \
printf("GPU Time used:  %3.1f ms\n", elapsedTime);\
checkCudaErrors(hipEventDestroy(start));\
checkCudaErrors(hipEventDestroy(stop));}


#define START_CPU {\
double start = omp_get_wtime();

#define END_CPU \
double end = omp_get_wtime();\
double duration = end - start;\
printf("CPU Time used: %3.1f ms\n", duration * 1000);}


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const double* A, const double* B, double* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = cos(A[i]) / sin(B[i]);
    }
}

/**
 * Host main routine
 */
int
main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 100*1024*1024;
    size_t size = numElements * sizeof(double);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the device input vector A
    double* d_A = NULL;
    err = hipHostMalloc((void**)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("A ok!");

    // Allocate the device input vector B
    double* d_B = NULL;
    err = hipHostMalloc((void**)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("B ok!");

    // Initialize the pinned memory input vectors
    for (int i = 0; i < numElements; ++i)
    {
        d_A[i] = rand() / (double)RAND_MAX;
        d_B[i] = rand() / (double)RAND_MAX;
    }

    // Allocate the device output vector C
    double* d_C = NULL;
    err = hipHostMalloc((void**)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    else
        printf("C ok!");

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    START_GPU
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    END_GPU

    START_GPU
    vectorAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C, numElements);
    END_GPU

    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    START_CPU
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(cos(d_A[i])/sin( d_B[i]) - d_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    END_CPU

    printf("Test PASSED\n");

    // Free device global memory
    err = hipHostFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipHostFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipHostFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

